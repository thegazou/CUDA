#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <limits.h>
#include <Indice1D.h>
#include "Indice2D.h"
#include "Indice1D.h"
#include "cudaTools.h"
// Each thread gets same seed, a different sequence number
// no offset
__global__ void createGenerator(hiprandState* tabDevGeneratorGM, int deviceId)
    {
// Customisation du generator:
// Proposition, au lecteur de faire mieux !
// Contrainte : Doit etre différent d'un GPU à l'autre
// Contrainte : Doit etre différent dun thread à lautre
    const int TID = Indice1D::tid();
    int deltaSeed = deviceId * INT_MAX / 10000;
    int deltaSequence = deviceId * 100;
    int deltaOffset = deviceId * 100;
    int seed = 1234 + deltaSeed;
    int sequenceNumber = TID + deltaSequence;
    int offset = deltaOffset;
    hiprand_init(seed, sequenceNumber, offset, &tabDevGeneratorGM[TID]);

    }
