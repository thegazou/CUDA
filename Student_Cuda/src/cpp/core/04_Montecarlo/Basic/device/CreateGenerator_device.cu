#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <limits.h>
#include <Indice1D.h>
#include "Indice2D.h"
#include "Indice1D.h"
#include "cudaTools.h"
__global__ void createGenerator(hiprandState* tabDevGeneratorGM, int deviceId)
    {
    const int TID = Indice1D::tid();
    int deltaSeed = deviceId * INT_MAX / 10000;
    int deltaSequence = deviceId * 100;
    int deltaOffset = deviceId * 100;
    int seed = 1234 + deltaSeed;
    int sequenceNumber = TID + deltaSequence;
    int offset = deltaOffset;
    hiprand_init(seed, sequenceNumber, offset, &tabDevGeneratorGM[TID]);
    }
