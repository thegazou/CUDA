#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <limits.h>
#include <Indice1D.h>
#include "Indice2D.h"
#include "Indice1D.h"
#include "cudaTools.h"
// no offset
__global__ void createGenerator2(hiprandState* tabDevGeneratorGM, int deviceId)
    {
    const int TID = Indice1D::tid();
    int deltaSeed = deviceId * INT_MAX / 10000;
    int deltaSequence = deviceId * 100;
    int deltaOffset = deviceId * 100;
    int seed = 1234 + deltaSeed;
    int sequenceNumber = TID + deltaSequence;
    int offset = deltaOffset;
    hiprand_init(seed, sequenceNumber, offset, &tabDevGeneratorGM[TID]);
    }
