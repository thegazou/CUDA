#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "IndiceTools_GPU.h"
#include "RipplingMath.h"

using namespace gpu;

// Attention : 	Choix du nom est impotant!
//		VagueDevice.cu et non Vague.cu
// 		Dans ce dernier cas, probl�me de linkage, car le nom du .cu est le meme que le nom d'un .cpp (host)
//		On a donc ajouter Device (ou n'importequoi) pour que les noms soient diff�rents!

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void rippling(uchar4* ptrDevPixels, uint w, uint h, float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void rippling(uchar4* ptrDevPixels, uint w, uint h, float t)
    {
    RipplingMath ripplingMath = RipplingMath(w);

    int i, j;
    const int WH = w * h;
    const int NB_THREAD = Indice2D::nbThread();
    const int TID = Indice2D::tid();
    int s = TID;

    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j);

	ripplingMath.colorIJ(&ptrDevPixels[s], (float) i, (float) j, t);

	s += NB_THREAD;
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

